#include"hip/hip_runtime.h"

#include"hiprand/hiprand_kernel.h"// this lib shoulb be included
#include<ctime>
#include<iostream>
#include<random>

using namespace std;
#define TAKE_BIT(x, pos) (((x) >> (pos)) & 0x1)


//v2 26r
/*__constant__ int version = 2;
__constant__ int N2=1024;
int hN2=1024;

int hN1=512;
__constant__ int N1=512;
__constant__ int r=6;
__constant__ unsigned char dk1[32] = {00,0x06, 00,0x20, 00,00, 00,0xd0, 
                                      00,0x0b, 00,0x60, 00,00, 00,0x70, 
                                      00,0x0a, 00,0x90, 00,00, 00,0x50, 
                                      00,0x03, 00,0x10, 00,00, 00,0x60};
__constant__ unsigned char dk2[32] = {00,00, 00,00, 00,00, 0x02,00, 
                                      00,00, 00,00, 00,00, 0x0f,00, 
                                      00,00, 00,00, 00,00, 0x0e,00, 
                                      00,00, 00,00, 00,00, 0x0e,00};
__constant__ unsigned char dp[8] = {00,0x09, 00,00, 00,00, 00,0x09}; 
__constant__ unsigned char dc[8] = {00,00, 00,00, 00,00, 00,00};*/


//v1 30r
__constant__ int version = 1;
__constant__ int N2=1024*8;
int hN2=1024*8;

int hN1=512*8;
__constant__ int N1=512*8;

__constant__ int r=5;
__constant__ unsigned char dk1[32] = {00,00, 00,00, 00,00, 00,0x01, 
                                      00,00, 00,00, 00,00, 00,0x08, 
                                      00,00, 00,00, 00,00, 00,00, 
                                      00,00, 00,00, 00,00, 00,0x08};
__constant__ unsigned char dk2[32] = {00,00, 00,00, 00,00, 0x01,00, 
                                      00,00, 00,00, 00,00, 0x06,00, 
                                      00,00, 00,00, 00,00, 00,00, 
                                      00,00, 00,00, 00,00, 0x07,00};
__constant__ unsigned char dp[8] = {00,00, 00,00, 0x10,00, 00,00}; 
__constant__ unsigned char dc[8] = {00,00, 0x20,00, 00,0x02, 00,0x20};




// 4-bit Sbox
__constant__ unsigned char sbox_4[16] = {12,6,9,0,1,10,2,11,3,8,5,13,4,14,7,15};
__constant__ unsigned char sbox_4_inv[16] = {3,4,6,8,12,10,1,14,9,2,5,7,0,11,13,15};


// ShiftAndSwitchRows permutation
__constant__ unsigned char P[16] = {0,1,2,3,7,4,5,6,10,11,8,9,13,14,15,12};
__constant__ unsigned char P_inv[16] = {0,1,2,3,5,6,7,4,10,11,8,9,15,12,13,14};

// Tweakey permutation
__constant__ unsigned char TWEAKEY_P[16] = {9,15,8,13,10,14,12,11,0,1,2,3,4,5,6,7};
__constant__ unsigned char TWEAKEY_P_inv[16] = {8,9,10,11,12,13,14,15,2,0,4,7,6,3,5,1};

// round constants
__constant__ unsigned char RC[62] = {
		0x01, 0x03, 0x07, 0x0F, 0x1F, 0x3E, 0x3D, 0x3B, 0x37, 0x2F,
		0x1E, 0x3C, 0x39, 0x33, 0x27, 0x0E, 0x1D, 0x3A, 0x35, 0x2B,
		0x16, 0x2C, 0x18, 0x30, 0x21, 0x02, 0x05, 0x0B, 0x17, 0x2E,
		0x1C, 0x38, 0x31, 0x23, 0x06, 0x0D, 0x1B, 0x36, 0x2D, 0x1A,
		0x34, 0x29, 0x12, 0x24, 0x08, 0x11, 0x22, 0x04, 0x09, 0x13,
		0x26, 0x0c, 0x19, 0x32, 0x25, 0x0a, 0x15, 0x2a, 0x14, 0x28,
		0x10, 0x20};

__device__ void AddKey(unsigned char state[4][4], unsigned char keyCells[4][4][4])
{
    int i, j, k;
    unsigned char pos;
    unsigned char keyCells_tmp[4][4][4];


    for(i = 0; i <= 1; i++)
    {
        for(j = 0; j < 4; j++)
        {
            state[i][j] ^= keyCells[0][i][j];
            state[i][j] ^= keyCells[1][i][j] ^ keyCells[2][i][j] ^ keyCells[3][i][j];
        }
    }


    for(k = 0; k <4; k++){
        for(i = 0; i < 4; i++){
            for(j = 0; j < 4; j++){     
                pos=TWEAKEY_P[j+4*i];
                keyCells_tmp[k][i][j]=keyCells[k][pos>>2][pos&0x3];
            }
        }
    }

    for(k = 0; k <4; k++){
        for(i = 0; i <= 1; i++){
            for(j = 0; j < 4; j++){
                if (k==1)
                {
                    keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]<<1)&0xE)^((keyCells_tmp[k][i][j]>>3)&0x1)^((keyCells_tmp[k][i][j]>>2)&0x1);
                    
                }
                else if (k==2)
                {
                    keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]>>1)&0x7)^((keyCells_tmp[k][i][j])&0x8)^((keyCells_tmp[k][i][j]<<3)&0x8);
                }
                else if (k==3)
                {
                    //v1
                    if(version == 1)
                        keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]<<1)&0xC)|(((TAKE_BIT(keyCells_tmp[k][i][j],0)^TAKE_BIT(keyCells_tmp[k][i][j],2))<<1)&0x2)|(TAKE_BIT(keyCells_tmp[k][i][j],1)^TAKE_BIT(keyCells_tmp[k][i][j],2)^TAKE_BIT(keyCells_tmp[k][i][j],3));
                    //v2
                    if(version == 2)
                        keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]<<2)&0xC)|(((TAKE_BIT(keyCells_tmp[k][i][j],3)^TAKE_BIT(keyCells_tmp[k][i][j],2))<<1)&0x2)|(TAKE_BIT(keyCells_tmp[k][i][j],1)^TAKE_BIT(keyCells_tmp[k][i][j],2));
                }
            }
        }
    }

    for(k = 0; k <4; k++){
        for(i = 0; i < 4; i++){
            for(j = 0; j < 4; j++){
                keyCells[k][i][j]=keyCells_tmp[k][i][j];
            }
        }
    }
}


__device__ void AddKey_inv(unsigned char state[4][4], unsigned char keyCells[3][4][4])
{
    int i, j, k;
    unsigned char pos;
    unsigned char keyCells_tmp[4][4][4];
    for(k = 0; k <4; k++){
        for(i = 0; i < 4; i++){
            for(j = 0; j < 4; j++){                
                pos=TWEAKEY_P_inv[j+4*i];
                keyCells_tmp[k][i][j]=keyCells[k][pos>>2][pos&0x3];
            }
        }
    }

    for(k = 0; k <4; k++){
        for(i = 2; i <= 3; i++){
            for(j = 0; j < 4; j++){
           
                if (k==1)
                {
                    keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]>>1)&0x7)^((keyCells_tmp[k][i][j]<<3)&0x8)^((keyCells_tmp[k][i][j])&0x8);                   
                }
                else if (k==2)
                {
                    keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]<<1)&0xE)^((keyCells_tmp[k][i][j]>>3)&0x1)^((keyCells_tmp[k][i][j]>>2)&0x1);                   
                }
                else if (k==3)
                {
                    //v1 inv
                    if(version == 1)
                        keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]>>1)&0x7)^(TAKE_BIT(keyCells_tmp[k][i][j],3)&0x1)^(((TAKE_BIT(keyCells_tmp[k][i][j],0)^TAKE_BIT(keyCells_tmp[k][i][j],2)^TAKE_BIT(keyCells_tmp[k][i][j],3))<<3)&0x8);
                    //v2 inv
                    if(version == 2)
                        keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]>>2)&0x3)^(((TAKE_BIT(keyCells_tmp[k][i][j],3)^TAKE_BIT(keyCells_tmp[k][i][j],1)^TAKE_BIT(keyCells_tmp[k][i][j],0))<<3)&0x8)^(((TAKE_BIT(keyCells_tmp[k][i][j],0)^TAKE_BIT(keyCells_tmp[k][i][j],3))<<2)&0x4);
                }
            }
        }
    }

    for(k = 0; k <4; k++){
        for(i = 0; i < 4; i++){
            for(j = 0; j < 4; j++){
                keyCells[k][i][j]=keyCells_tmp[k][i][j];
            }
        }
    }

    for(i = 0; i <= 1; i++)
    {
        for(j = 0; j < 4; j++)
        {
            state[i][j] ^= keyCells[0][i][j];
            state[i][j] ^= keyCells[1][i][j] ^ keyCells[2][i][j] ^ keyCells[3][i][j];
        }
    }
}


__device__ void AddConstants(unsigned char state[4][4], int r)
{
	state[0][0] ^= (RC[r] & 0xf);
	state[1][0] ^= ((RC[r]>>4) & 0x3);
	state[2][0] ^= 0x2;
}

__device__ void SubCell4(unsigned char state[4][4])
{
	int i,j;
	for(i = 0; i < 4; i++)
		for(j = 0; j <  4; j++)
			state[i][j] = sbox_4[state[i][j]];
}


__device__ void SubCell4_inv(unsigned char state[4][4])
{
	int i,j;
	for(i = 0; i < 4; i++)
		for(j = 0; j <  4; j++)
			state[i][j] = sbox_4_inv[state[i][j]];
}


__device__ void ShiftRows(unsigned char state[4][4])
{
	int i, j, pos;

	unsigned char state_tmp[4][4];
    for(i = 0; i < 4; i++)
    {
        for(j = 0; j < 4; j++)
        {
            pos=P[j+4*i];
            state_tmp[i][j]=state[pos>>2][pos&0x3];
        }
    }

    for(i = 0; i < 4; i++)
    {
        for(j = 0; j < 4; j++)
        {
            state[i][j]=state_tmp[i][j];
        }
    }
}

__device__ void ShiftRows_inv(unsigned char state[4][4])
{
	int i, j, pos;

	unsigned char state_tmp[4][4];
    for(i = 0; i < 4; i++)
    {
        for(j = 0; j < 4; j++)
        {
            pos=P_inv[j+4*i];
            state_tmp[i][j]=state[pos>>2][pos&0x3];
        }
    }

    for(i = 0; i < 4; i++)
    {
        for(j = 0; j < 4; j++)
        {
            state[i][j]=state_tmp[i][j];
        }
    }
}


__device__ void MixColumn(unsigned char state[4][4])
{
	int j;
    unsigned char temp;

	for(j = 0; j < 4; j++){
        state[1][j]^=state[2][j];
        state[2][j]^=state[0][j];
        state[3][j]^=state[2][j];

        temp=state[3][j];
        state[3][j]=state[2][j];
        state[2][j]=state[1][j];
        state[1][j]=state[0][j];
        state[0][j]=temp;
	}
}


__device__ void MixColumn_inv(unsigned char state[4][4])
{
	int j;
    unsigned char temp;

	for(j = 0; j < 4; j++){
        temp=state[3][j];
        state[3][j]=state[0][j];
        state[0][j]=state[1][j];
        state[1][j]=state[2][j];
        state[2][j]=temp;

        state[3][j]^=state[2][j];
        state[2][j]^=state[0][j];
        state[1][j]^=state[2][j];
	}
}

__device__ void ENC(unsigned char* input, const unsigned char* userkey, int r)
{
	unsigned char state[4][4];
	unsigned char keyCells[4][4][4];
	int i;

	for(i=0; i<4; i++)
		for(int j=0; j<4; j++)
			for(int k=0; k<4; k++)
				keyCells[i][j][k]=0;                                 
	for(i = 0; i < 16; i++) {
            if(i&1)
            {
                state[i>>2][i&0x3] = input[i>>1]&0xF;
                keyCells[0][i>>2][i&0x3] = userkey[i>>1]&0xF;
                keyCells[1][i>>2][i&0x3] = userkey[(i+16)>>1]&0xF;                
                keyCells[2][i>>2][i&0x3] = userkey[(i+32)>>1]&0xF;
                keyCells[3][i>>2][i&0x3] = userkey[(i+48)>>1]&0xF;
            }
            else
            {
                state[i>>2][i&0x3] = (input[i>>1]>>4)&0xF;
                keyCells[0][i>>2][i&0x3] = (userkey[i>>1]>>4)&0xF;
                keyCells[1][i>>2][i&0x3] = (userkey[(i+16)>>1]>>4)&0xF;
                keyCells[2][i>>2][i&0x3] = (userkey[(i+32)>>1]>>4)&0xF;
                keyCells[3][i>>2][i&0x3] = (userkey[(i+48)>>1]>>4)&0xF;
            }
    }


	for(i = 0; i < r; i++){
        SubCell4(state);

 
        AddConstants(state, i);
           
        AddKey(state, keyCells);
         
        ShiftRows(state);
            
        MixColumn(state);
           
		
	}

	
        for(i = 0; i < 8; i++)
            input[i] = ((state[(2*i)>>2][(2*i)&0x3] & 0xF) << 4) | (state[(2*i+1)>>2][(2*i+1)&0x3] & 0xF);


}


__device__ void DEC(unsigned char* input, const unsigned char* userkey, int r)
{
	unsigned char state[4][4];
	unsigned char dummy[4][4]={{0}};
	unsigned char keyCells[4][4][4];
	int i;

    	for(i=0; i<4; i++)
		for(int j=0; j<4; j++)
			for(int k=0; k<4; k++)
				keyCells[i][j][k]=0;
	for(i = 0; i < 16; i++) {
 
            if(i&1)
            {
                state[i>>2][i&0x3] = input[i>>1]&0xF;
                keyCells[0][i>>2][i&0x3] = userkey[i>>1]&0xF;
                keyCells[1][i>>2][i&0x3] = userkey[(i+16)>>1]&0xF;
                keyCells[2][i>>2][i&0x3] = userkey[(i+32)>>1]&0xF;
                keyCells[3][i>>2][i&0x3] = userkey[(i+48)>>1]&0xF;
            }
            else
            {
                state[i>>2][i&0x3] = (input[i>>1]>>4)&0xF;
                keyCells[0][i>>2][i&0x3] = (userkey[i>>1]>>4)&0xF;
                keyCells[1][i>>2][i&0x3] = (userkey[(i+16)>>1]>>4)&0xF;
                keyCells[2][i>>2][i&0x3] = (userkey[(i+32)>>1]>>4)&0xF;
                keyCells[3][i>>2][i&0x3] = (userkey[(i+48)>>1]>>4)&0xF;
            }
        
    }

    for(i = r-1; i >=0 ; i--){
        AddKey(dummy, keyCells);
    }

    

	for(i = r-1; i >=0 ; i--){
        MixColumn_inv(state);
            
        ShiftRows_inv(state);
           
        AddKey_inv(state, keyCells);
           
        AddConstants(state, i);
            
        SubCell4_inv(state);

            
	}

        for(i = 0; i < 8; i++)
            input[i] = ((state[(2*i)>>2][(2*i)&0x3] & 0xF) << 4) | (state[(2*i+1)>>2][(2*i+1)&0x3] & 0xF);
 

}

//-------------------generate random numbers-------//
__device__ float generate(hiprandState *globalState, int ind)
{
	hiprandState localState = globalState[ind];
	float RANDOM = hiprand_uniform(&localState);// uniform distribution
	globalState[ind] = localState;
	return RANDOM;
}

__global__ void setup_kernel(hiprandState *state, unsigned long seed)
{
	int ix = threadIdx.x + blockIdx.x*blockDim.x;
	int iy = threadIdx.y + blockIdx.y*blockDim.y;
	int idx = iy * blockDim.x*gridDim.x + ix;
	
	hiprand_init(seed, idx, 0, &state[idx]);// initialize the state
}

//-------------This is our kernel function where the random numbers generated------//
__global__ void our_kernel(hiprandState *globalState,int *devNum)
{
	int ix = threadIdx.x + blockIdx.x*blockDim.x;
	int iy = threadIdx.y + blockIdx.y*blockDim.y;
	int idx = iy * blockDim.x*gridDim.x + ix;

	int i,j,jj;
	int num=0;
	unsigned char p1[8],p2[8];
	unsigned char c3[8],c4[8];
	unsigned char k1[32], k2[32], k3[32], k4[32];
	bool flag;
	
	int k;
	// randomly choose k1
	for(i = 0; i < 32; i++) 
	{
		k = generate(globalState, idx) * 100000;
		k1[i] = k & 0xff;
		//printf("k1[%d]=%d\n", i,k1[i]);
	}	
	for(i = 0; i < 32; i++) 
		k2[i] = k1[i]^dk1[i];	
	for(i = 0; i < 32; i++) 
		k3[i] = k1[i]^dk2[i];	
	for(i = 0; i < 32; i++) 
		k4[i] = k2[i]^dk2[i];
	
	for (j=0; j<N1; j++)
	{
	for (jj=0; jj<N2; jj++)
	{
		//printf("[%d]=%d\n",idx,j);
		// randomly choose p1
		for(i = 0; i < 8; i++) 
		{
			k = generate(globalState, idx) * 100000;
			p1[i] = k & 0xff;
			//printf("P1[%d]=%d\n", i,p1[i]);	
		}
		// derive p2
		for(i = 0; i < 8; i++) 
			p2[i] = p1[i]^dp[i];	

		ENC(p1, k1, r);
		ENC(p2, k2, r);
		
		// derive c3
		for(i = 0; i < 8; i++) 
			c3[i] = p1[i]^dc[i];
		// derive c4
		for(i = 0; i < 8; i++) 
			c4[i] = p2[i]^dc[i];
		DEC(c3, k3, r);
		DEC(c4, k4, r);
		flag = 1;
		for(i = 0; i < 8; i++)
		{
			//printf("c3=%d, c4=%d\n", c3[i],c4[i]);	
			if ((c3[i]^c4[i]) != dp[i])
				flag = 0;
		}
		if (flag) 
		{
			num ++; 
		}		
		//printf("%d\n", k);
	}
	}
	
	devNum[idx]=num;
	//printf("%d\n",devNum[idx]);
}

int main()
{
	int blockx = 256;
	int blocky = 1;
	dim3 block(blockx, blocky);

	int gridx = 1;
	int gridy = 128;
	dim3 grid(gridx,gridy); 

	int N = gridx*gridy*blockx*blocky;// the number of states
	int *hostNum = (int *)malloc(N * sizeof(int));
	double sum=0;

	hipEvent_t start, stop;
	float elapsedTime = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	//--------------------//
	hiprandState* devStates;
	int* devNum;
	hipError_t err = hipSuccess;
	err=hipMalloc(&devStates, N * sizeof(hiprandState));
	err=hipMalloc((void **)&devNum, N * sizeof(int));
	if(err!=hipSuccess)
    	{
        	printf("the hipMalloc on GPU is failed\n");
        	return 1;
    	}
	printf("SUCCESS\n");

	srand(time(0));
	int seed = rand();

	//  Initialize the states
	setup_kernel <<<grid, block>>> (devStates, seed);

	our_kernel <<<grid, block >>> (devStates,devNum);
 	
	err = hipGetLastError();

    	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
   	}
	
	err=hipMemcpy(hostNum,devNum,N*sizeof(int),hipMemcpyDeviceToHost);
	
	if (err != hipSuccess)
   	{
        	fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

	for(int i=0;i<N;i++)
		sum += hostNum[i];
	sum = double(N)/sum;
	//cout <<  log(sum)/log(2.0) << endl;
	cout <<  (-log(sum)/log(2.0)-log(hN1)/log(2.0)-log(hN2)/log(2.0)) << endl;

	hipFree(devNum);
	hipFree(devStates);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	cout << (elapsedTime/1000) <<'s'<< endl; 

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceReset();
	return 0;
}
